#include "hip/hip_runtime.h"
/*  Vector percent change: C = log(A[x]/A[x-1])
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

//#include "cutil.h"

// includes, kernels
#include "vectorPercentChange_kernel.cu"

#define MAXLINE 100000

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float* C, const float* A, unsigned int N);

Vector AllocateDeviceVector(const Vector V);
Vector AllocateVector(int size, int init);
void CopyToDeviceVector(Vector Vdevice, const Vector Vhost);
void CopyFromDeviceVector(Vector Vhost, const Vector Vdevice);
int ReadFile(Vector* V, char* file_name);
void WriteFile(Vector V, char* file_name);

void VectorPercentChange(const Vector A, Vector C);
void printVector(Vector M);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

	// Vectors for the program
	Vector A;
	Vector C;
	// Number of elements in the vectors
	unsigned int size_elements = VSIZE;
	int errorA = 0;
	
	srand(2012);
	
	// Check command line for input vector files
	printf("argc %d\n",argc);
	if(argc == 1 || argc == 2) 
	{
		// No inputs provided
		// Allocate and initialize the vectors
		A  = AllocateVector(VSIZE, 1);
		C  = AllocateVector(VSIZE, 0);
	}
	else
	{
		// Inputs provided
		// Allocate and read source vectors from disk
		A  = AllocateVector(VSIZE, 0);		
		C  = AllocateVector(VSIZE, 0);
		errorA = ReadFile(&A, argv[2]);
		printVector(A);
		// check for read errors
		if(errorA != size_elements)
		{
			printf("Error reading input files %d, %d\n", errorA);
			return 1;
		}
	}
	
	// A + B on the device
    VectorPercentChange(A, C);
    // compute the vector addition on the CPU for comparison
    Vector reference = AllocateVector(size_elements, 0);
    computeGold(reference.elements,A.elements,size_elements);    
    // check if the device result is equivalent to the expected solution
    //CUTBoolean res = cutComparefe(reference.elements, C.elements, 
	//								size_elements, 0.0001f);
    unsigned res = 1;
    for (unsigned i = 0; i < size_elements; i++)
        if (abs(reference.elements[i] - C.elements[i]) > 0.0001f)
            res = 0;

    printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");
    
    // output result if output file is requested
    if(argc == 3)
    {
		WriteFile(C, argv[1]);
	}
	else if(argc == 2)
	{
	    WriteFile(C, argv[1]);
	}    

	// Free host matrices
    free(A.elements);
    A.elements = NULL;
    free(C.elements);
    C.elements = NULL;
	return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void VectorPercentChange(const Vector A, Vector C)
{
	//Interface host call to the device kernel code and invoke the kernel
   	//hipError_t cuda_ret;
	Vector d_A, d_C;
	dim3 dim_grid, dim_block;
    // steps:
    
    // 1. allocate device vectors d_A and d_C with length same as input vector
    d_A = AllocateDeviceVector(A);
    d_C = AllocateDeviceVector(C);
    // 2. copy A to d_A,
    CopyToDeviceVector(d_A, A);
    // 3. launch kernel to compute d_C = d_An[1]/d_An[x-1]
    
    dim_block.x = BLOCK_SIZE; dim_block.y = dim_block.z = 1;
    dim_grid.x = A.length / dim_block.x;
    if(A.length % dim_block.x != 0) dim_grid.x++;
    dim_grid.y = 1;
    dim_grid.z = 1;
    VectorPercentChangeKernel<<<dim_grid, dim_block>>>(d_A, d_C);
    
    // 4. copy d_C back to host vector C
    CopyFromDeviceVector(C, d_C);
    // 5. free device vectors d_A, d_B, d_C
    hipFree(d_A.elements);
    hipFree(d_C.elements);
    
}

// Allocate a device vector of same size as V.
Vector AllocateDeviceVector(const Vector V)
{
    Vector Vdevice = V;
    int size = V.length * sizeof(float);
    hipError_t cuda_ret = hipMalloc((void**)&Vdevice.elements, size);
    if(cuda_ret != hipSuccess) {
        printf("Unable to allocate device memory");
        exit(0);
    }
    return Vdevice;
}

// Allocate a vector of dimensions length
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
Vector AllocateVector(int length, int init)
{
    Vector V;
    V.length = length;
    V.elements = NULL;
		
	V.elements = (float*) malloc(length*sizeof(float));

	for(unsigned int i = 0; i < V.length; i++)
	{
		V.elements[i] = (init == 0) ? (0.0f) : (rand() / (float)RAND_MAX);
	}
    return V;
}	

// Copy a host vector to a device vector.
void CopyToDeviceVector(Vector Vdevice, const Vector Vhost)
{
    int size = Vhost.length * sizeof(float);
    Vdevice.length = Vhost.length;
    hipMemcpy(Vdevice.elements, Vhost.elements, size, 
					hipMemcpyHostToDevice);
}

// Copy a device vector to a host vector.
void CopyFromDeviceVector(Vector Vhost, const Vector Vdevice)
{
    int size = Vdevice.length * sizeof(float);
    hipMemcpy(Vhost.elements, Vdevice.elements, size, 
					hipMemcpyDeviceToHost);
}

// Read a floating point vector in from file
int ReadFile(Vector* V, char* file_name)
{
	unsigned int data_read = VSIZE;
	FILE* input = fopen(file_name, "r");
    char vector_string[MAXLINE];
    fgets(vector_string, MAXLINE, input);
    char* part = strtok(vector_string, " ");
    for (unsigned i = 0; i < VSIZE; i++) {
        V->elements[i] = atof(part);
        part = strtok(NULL, " ");
    }
	return data_read;
}

// Write a floating point vector to file
void WriteFile(Vector V, char* file_name)
{
    FILE* output = fopen(file_name, "w");
    for (unsigned i = 0; i < VSIZE; i++) {
        fprintf(output, "%f ", V.elements[i]);
    }
}

void printVector(Vector M){   
    for (int i = 0; i < M.length; i++){
		printf("%f|", M.elements[i]);
	}
	printf("\n");
}
